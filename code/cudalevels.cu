#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <unistd.h>
#include <vector>
#include <array>
#include <thread>
#include <cmath>
#include <mutex>
#include <atomic>
#include <barrier>
#include "constants.hpp"
#include "helpers.hpp"

__global__ void update_leaf_nodes(int* ST, const int* device_ops, const int batch_start, const int batch_size, const int array_size){
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_stride = gridDim.x * blockDim.x;
    for (int op_i = thread_id + batch_start; op_i < batch_start + batch_size; op_i += thread_stride){
        int i = device_ops[3 * op_i + 1];
        int increment = device_ops[3 * op_i + 2];

        int u = i + array_size - 1;
        atomicAdd(&ST[u],increment);
    }
}

__global__ void update_interior_nodes(int* ST, const int* device_ops, const int batch_start, const int batch_size, const int array_size, const int level){
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_stride = gridDim.x * blockDim.x;

    int num_nodes = 1 << level;
    int level_offset = num_nodes - 1;
    for (int level_index = thread_id+level_offset; level_index < num_nodes+level_offset; level_index += thread_stride){
        ST[level_index] = ST[2*level_index+1] + ST[2*level_index+2];
    }
}

__device__ int deviceComputeSum(int u, int i, int j, int L, int R, const int* ST) {
    if (i <= L && R <= j) {
        return ST[u];
    }
    else {
        int mid = (L + R)/2;
        if (i >= mid){
            return deviceComputeSum(2*u+2,i,j,mid,R,ST);
        }
        else if (j <= mid){
            return deviceComputeSum(2*u+1,i,j,L,mid,ST);
        }
        else{
            return deviceComputeSum(2*u+1,i,j,L,mid,ST) + deviceComputeSum(2*u+2,i,j,mid,R,ST);
        }
    }
}

__global__ void process_queries(const int* ST, const int* device_ops, int2* device_query_results, int batch_start, int batch_size, int array_size) {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id >= batch_size) return;

    int op_i = batch_start + thread_id;
    int i = device_ops[3*op_i+1];
    int j = device_ops[3*op_i+2];

    int sum = deviceComputeSum(0, i, j, 0, array_size, ST);
    device_query_results[thread_id] = make_int2(op_i, sum);
}

std::chrono::duration<double> cudalevels_total_query_time(0.0);
std::chrono::duration<double> cudalevels_total_levels_time(0.0);
std::chrono::duration<double> cudalevels_total_update_time(0.0);

void runLevelsCudaImplementation(const int num_ops, const int num_query, const int num_update, const int array_size, const int ST_size, const std::vector<std::array<int, 3>>& ops,std::vector<std::array<int, 2>>& query_results){
    std::vector<int> batch_starts;
    batch_starts.push_back(0);
    int prev_type = ops[0][0];
    int type = -1;
    for (int op_i = 1; op_i < num_ops; op_i ++){
        type = ops[op_i][0];
        if (type != prev_type){
            batch_starts.push_back(op_i);
        }
        prev_type = type;
    }
    int num_batches = batch_starts.size();

    int max_batch_size = 0;
    for (int b = 0; b < num_batches; ++b) {
        int start = batch_starts[b];
        int end   = (b+1 < (int)batch_starts.size() ? batch_starts[b+1] : num_ops);
        max_batch_size = std::max(max_batch_size, end - start);
    }

    int* device_ST;
    int ST_byte_size = ST_size * sizeof(int);
    hipMalloc(&device_ST,ST_byte_size);
    hipMemset(device_ST,0,ST_byte_size);

    int* device_ops;
    int ops_byte_size = num_ops * sizeof(std::array<int,3>);
    hipMalloc(&device_ops,ops_byte_size);
    hipMemcpy(device_ops,&ops[0],ops_byte_size,hipMemcpyHostToDevice);

    int2* device_query_results_buffer;
    hipMalloc(&device_query_results_buffer, max_batch_size * sizeof(int2));

    int2* host_query_results_buffer;
    hipHostMalloc(&host_query_results_buffer, max_batch_size * sizeof(int2));
    
    int threads_per_block = 256;

    int last_level_ind = std::log2(array_size);

    int query_offset = 0;
    for(int batch_iter = 0; batch_iter < num_batches; batch_iter++){
        int batch_start = batch_starts[batch_iter];
        int batch_end = (batch_iter == num_batches-1) ? num_ops : batch_starts[batch_iter+1];
        int batch_type = ops[batch_start][0];
        int batch_size = batch_end - batch_start;

        int num_blocks = (batch_size + threads_per_block - 1)/threads_per_block;

        const auto batch_start_time = std::chrono::steady_clock::now();

        if(batch_type == UPDATE){
            update_leaf_nodes<<<num_blocks,threads_per_block>>>(device_ST,device_ops,batch_start,batch_size,array_size);
            const auto levels_start_time = std::chrono::steady_clock::now();
            for (int level = last_level_ind - 1; level >= 0; level--){
                update_interior_nodes<<<num_blocks,threads_per_block>>>(device_ST,device_ops,batch_start,batch_size,array_size,level);
            }
            auto levels_end_time = std::chrono::steady_clock::now();
            std::chrono::duration<double> elapsed = levels_end_time - levels_start_time;
            cudalevels_total_levels_time += elapsed;
        }
        else if(batch_type == QUERY){
            process_queries<<<num_blocks, threads_per_block>>>(
                device_ST,
                device_ops,
                device_query_results_buffer,
                batch_start,
                batch_size,
                array_size
            );

            hipMemcpy(
                host_query_results_buffer,
                device_query_results_buffer,
                batch_size * sizeof(int2),
                hipMemcpyDeviceToHost
            );
            for (int t = 0; t < batch_size; ++t) {
                query_results[query_offset + t] = {host_query_results_buffer[t].x, host_query_results_buffer[t].y};
            }
            query_offset += batch_size;
        }
        auto batch_end_time = std::chrono::steady_clock::now();
        std::chrono::duration<double> elapsed = batch_end_time - batch_start_time;
        if (batch_type == UPDATE) {
            cudalevels_total_update_time += elapsed;
        } else {
            cudalevels_total_query_time += elapsed;
        }
    }

    /* Timing code */
    std::cout << "Total query time: " << cudalevels_total_query_time.count() << " sec" << '\n';
    std::cout << "Total levels time: " << cudalevels_total_levels_time.count() << " sec" << '\n';
    std::cout << "Total update time: " << cudalevels_total_update_time.count() << " sec" << '\n';

    hipFree(device_ST);
    hipFree(device_ops);
    hipFree(device_query_results_buffer);
    hipHostFree(host_query_results_buffer);    
}

