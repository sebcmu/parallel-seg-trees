#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <unistd.h>
#include <vector>
#include <array>
#include <thread>
#include <cmath>
#include <mutex>
#include <atomic>
#include <barrier>
#include "constants.hpp"
#include "helpers.hpp"

__global__ void update_leaf_nodes(int* ST, const int* device_ops, const int batch_start, const int batch_size, const int array_size){
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_stride = gridDim.x * blockDim.x;
    for (int op_i = thread_id + batch_start; op_i < batch_start + batch_size; op_i += thread_stride){
        int i = device_ops[3 * op_i + 1];
        int increment = device_ops[3 * op_i + 2];

        int u = i + array_size - 1;
        atomicAdd(&ST[u],increment);
    }
}

__global__ void update_interior_nodes(int* ST, const int* device_ops, const int batch_start, const int batch_size, const int array_size, const int level){
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_stride = gridDim.x * blockDim.x;

    int num_nodes = 1 << level;
    int level_offset = num_nodes - 1;
    for (int level_index = thread_id+level_offset; level_index < num_nodes+level_offset; level_index += thread_stride){
        ST[level_index] = ST[2*level_index+1] + ST[2*level_index+2];
    }
}

void runLevelsCudaImplementation(const int num_ops, const int array_size, const int ST_size, const std::vector<std::array<int, 3>>& ops,std::vector<std::array<int, 2>>& query_results){
    std::vector<int> batch_starts;
    batch_starts.push_back(0);
    int prev_type = ops[0][0];
    int type = -1;
    for (int op_i = 1; op_i < num_ops; op_i ++){
        type = ops[op_i][0];
        if (type != prev_type){
            batch_starts.push_back(op_i);
        }
        prev_type = type;
    }
    int num_batches = batch_starts.size();

    int ST_byte_size = ST_size * sizeof(int);

    int* device_ST;
    hipMalloc(&device_ST,ST_byte_size);
    hipMemset(device_ST,0,ST_byte_size);

    int* device_ops;
    int ops_byte_size = num_ops * sizeof(std::array<int,3>);
    hipMalloc(&device_ops,ops_byte_size);
    hipMemcpy(device_ops,&ops[0],ops_byte_size,hipMemcpyHostToDevice);

    std::vector<int> host_ST(ST_size);
    int threads_per_block = 256;

    int last_level_ind = std::log2(array_size);

    int query_offset = 0;
    for(int batch_iter = 0; batch_iter < num_batches; batch_iter++){
        int batch_start = batch_starts[batch_iter];
        int batch_end = (batch_iter == num_batches-1) ? num_ops : batch_starts[batch_iter+1];
        int batch_type = ops[batch_start][0];
        int batch_size = batch_end - batch_start;

        int num_blocks = (batch_size + threads_per_block - 1)/threads_per_block;

        if(batch_type == UPDATE){
            update_leaf_nodes<<<num_blocks,threads_per_block>>>(device_ST,device_ops,batch_start,batch_size,array_size);
            for (int level = last_level_ind - 1; level >= 0; level--){
                update_interior_nodes<<<num_blocks,threads_per_block>>>(device_ST,device_ops,batch_start,batch_size,array_size,level);
            }
        }
        else if(batch_type == QUERY){
            hipMemcpy(&host_ST[0],device_ST,ST_byte_size,hipMemcpyDeviceToHost);
            
            for (int op_i = batch_start; op_i < batch_end; op_i++){
                int i = ops[op_i][1];
                int j = ops[op_i][2];
                int query_answer = computeSum(0,i,j,0,array_size,host_ST);
                query_results[query_offset] = {op_i,query_answer};
                query_offset++;
            }
        }
    }
    hipFree(device_ST);
    hipFree(device_ops);
}

