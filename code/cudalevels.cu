#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <unistd.h>
#include <vector>
#include <array>
#include <thread>
#include <cmath>
#include <mutex>
#include <atomic>
#include <barrier>
#include "constants.hpp"
#include "helpers.hpp"

/* Function Structs to Define the Template for the Kernels*/
/* Need to uopdate these everytime a new combine_fn is added to main*/
struct CombineSum{
    __device__ static void combine_leaf_fn(int* st_address, int new_value){atomicAdd(st_address,new_value);}
    __device__ static int combine_interior_fn(int a, int b){return a+b;}
};
struct CombineMax{
    __device__ static void combine_leaf_fn(int* st_address, int new_value){
        int expected, desired;
        do{
            expected = *st_address;
            desired = max(expected,new_value);
        } while (atomicCAS(st_address,expected,desired) != expected);
    }
    __device__ static int combine_interior_fn(int a, int b){return max(a,b);}
};
struct CombineMin{
    __device__ static void combine_leaf_fn(int* st_address, int new_value){
        int expected, desired;
        do{
            expected = *st_address;
            desired = min(expected,new_value);
        } while (atomicCAS(st_address,expected,desired) != expected);
    }
    __device__ static int combine_interior_fn(int a, int b){return min(a,b);}
};

template <typename IntCombine>
__global__ void update_leaf_nodes(int* ST, const int* device_ops, const int batch_start, const int batch_size, const int array_size){
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_stride = gridDim.x * blockDim.x;
    for (int op_i = thread_id + batch_start; op_i < batch_start + batch_size; op_i += thread_stride){
        int i = device_ops[3 * op_i + 1];
        int increment = device_ops[3 * op_i + 2];

        int u = i + array_size - 1;
        IntCombine::combine_leaf_fn(&ST[u],increment);
    }
}

template <typename IntCombine>
__global__ void update_interior_nodes(int* ST, const int* device_ops, const int batch_start, const int batch_size, const int array_size, const int level){
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_stride = gridDim.x * blockDim.x;

    int num_nodes = 1 << level;
    int level_offset = num_nodes - 1;
    for (int level_index = thread_id+level_offset; level_index < num_nodes+level_offset; level_index += thread_stride){
        ST[level_index] = IntCombine::combine_interior_fn(ST[2*level_index+1],ST[2*level_index+2]);
    }
}

void runCudaLevelsImplementation(const std::vector<int>& batch_starts,const int num_ops, const int num_query, const int num_update, const int array_size, const int ST_size, const std::vector<std::array<int, 3>>& ops,std::vector<std::array<int, 2>>& query_results, const int combine_type, IntCombine combine_fn){
    int num_batches = batch_starts.size();

    int ST_byte_size = ST_size * sizeof(int);

    int* device_ST;
    hipMalloc(&device_ST,ST_byte_size);
    hipMemset(device_ST,0,ST_byte_size);

    int* device_ops;
    int ops_byte_size = num_ops * sizeof(std::array<int,3>);
    hipMalloc(&device_ops,ops_byte_size);
    hipMemcpy(device_ops,&ops[0],ops_byte_size,hipMemcpyHostToDevice);

    std::vector<int> host_ST(ST_size);
    int threads_per_block = 256;

    int last_level_ind = std::log2(array_size);
    int query_offset = 0;
    if (combine_type == COMBINE_SUM_FLAG){
        for(int batch_iter = 0; batch_iter < num_batches; batch_iter++){
            int batch_start = batch_starts[batch_iter];
            int batch_end = (batch_iter == num_batches-1) ? num_ops : batch_starts[batch_iter+1];
            int batch_type = ops[batch_start][0];
            int batch_size = batch_end - batch_start;
    
            int num_blocks = (batch_size + threads_per_block - 1)/threads_per_block;
    
            if(batch_type == UPDATE){
                update_leaf_nodes<CombineSum><<<num_blocks,threads_per_block>>>(device_ST,device_ops,batch_start,batch_size,array_size);
                for (int level = last_level_ind - 1; level >= 0; level--){
                    update_interior_nodes<CombineSum><<<num_blocks,threads_per_block>>>(device_ST,device_ops,batch_start,batch_size,array_size,level);
                }
            }
            else if(batch_type == QUERY){
                hipMemcpy(&host_ST[0],device_ST,ST_byte_size,hipMemcpyDeviceToHost);
                for (int op_i = batch_start; op_i < batch_end; op_i++){
                    int i = ops[op_i][1];
                    int j = ops[op_i][2];
                    int query_answer = computeSumCombine(0,i,j,0,array_size,host_ST,combine_fn,combine_type);
                    query_results[query_offset][OPERATION_INDEX] = op_i;
                    query_results[query_offset][QUERY_ANS] = query_answer;
                    query_offset++;
                }
            }
        }
    } else if (combine_type == COMBINE_MAX_FLAG){
        for(int batch_iter = 0; batch_iter < num_batches; batch_iter++){
            int batch_start = batch_starts[batch_iter];
            int batch_end = (batch_iter == num_batches-1) ? num_ops : batch_starts[batch_iter+1];
            int batch_type = ops[batch_start][0];
            int batch_size = batch_end - batch_start;
    
            int num_blocks = (batch_size + threads_per_block - 1)/threads_per_block;
    
            if(batch_type == UPDATE){
                update_leaf_nodes<CombineMax><<<num_blocks,threads_per_block>>>(device_ST,device_ops,batch_start,batch_size,array_size);
                for (int level = last_level_ind - 1; level >= 0; level--){
                    update_interior_nodes<CombineMax><<<num_blocks,threads_per_block>>>(device_ST,device_ops,batch_start,batch_size,array_size,level);
                }
            }
            else if(batch_type == QUERY){
                hipMemcpy(&host_ST[0],device_ST,ST_byte_size,hipMemcpyDeviceToHost);
                
                for (int op_i = batch_start; op_i < batch_end; op_i++){
                    int i = ops[op_i][1];
                    int j = ops[op_i][2];
                    // must fix!!
                    int query_answer = computeSumCombine(0,i,j,0,array_size,host_ST,combine_fn,combine_type);
                    query_results[query_offset] = {op_i,query_answer};
                    query_offset++;
                }
            }
        }
    }else if (combine_type == COMBINE_MIN_FLAG){
        for(int batch_iter = 0; batch_iter < num_batches; batch_iter++){
            int batch_start = batch_starts[batch_iter];
            int batch_end = (batch_iter == num_batches-1) ? num_ops : batch_starts[batch_iter+1];
            int batch_type = ops[batch_start][0];
            int batch_size = batch_end - batch_start;
    
            int num_blocks = (batch_size + threads_per_block - 1)/threads_per_block;
    
            if(batch_type == UPDATE){
                update_leaf_nodes<CombineMin><<<num_blocks,threads_per_block>>>(device_ST,device_ops,batch_start,batch_size,array_size);
                for (int level = last_level_ind - 1; level >= 0; level--){
                    update_interior_nodes<CombineMin><<<num_blocks,threads_per_block>>>(device_ST,device_ops,batch_start,batch_size,array_size,level);
                }
            }
            else if(batch_type == QUERY){
                hipMemcpy(&host_ST[0],device_ST,ST_byte_size,hipMemcpyDeviceToHost);
                
                for (int op_i = batch_start; op_i < batch_end; op_i++){
                    int i = ops[op_i][1];
                    int j = ops[op_i][2];
                    int query_answer = computeSumCombine(0,i,j,0,array_size,host_ST,combine_fn,combine_type);
                    query_results[query_offset] = {op_i,query_answer};
                    query_offset++;
                }
            }
        }
    }

    hipFree(device_ST);
    hipFree(device_ops);
}