#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <unistd.h>
#include <vector>
#include <array>
#include <thread>
#include <cmath>
#include <mutex>
#include <atomic>
#include <barrier>
#include "constants.hpp"

__global__ void process_update_batch(int* prefix_sum, const int* device_ops, const int batch_start, const int batch_size, const int prefix_sum_size){
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_id = thread_id/32;
    int warp_stride = (gridDim.x * blockDim.x + 31) / 32;
    int lane_id = threadIdx.x % 32;
    int lane_stride = 32;
    /* Each warp takes an update, uses lanes to quickly increment positions [i+1,prefix_sum_size) by x*/
    for (int i = warp_id + batch_start; i<batch_start+batch_size; i+=warp_stride){
        int start_index = device_ops[3 * i + 1] + 1;
        int increment = device_ops[3 * i + 2];
        
        /*Each lane takes an increment*/
        for (int j = start_index + lane_id; j < prefix_sum_size; j += lane_stride){
            atomicAdd(&prefix_sum[j],increment);
        }
    }
}

void runPrefixCudaImplementation(const int num_ops, const int array_size, const std::vector<std::array<int, 3>>& ops,std::vector<std::array<int, 2>>& query_results){
    std::vector<int> batch_starts;
    batch_starts.push_back(0);
    int prev_type = ops[0][0];
    int type = -1;
    for (int op_i = 1; op_i < num_ops; op_i ++){
        type = ops[op_i][0];
        if (type != prev_type){
            batch_starts.push_back(op_i);
        }
        prev_type = type;
    }
    int num_batches = batch_starts.size();

    /* PS[i] = prefix sum up to but not including i, so need array_size + 1 spaces to store all info (PS[0] = 0 trivially and does not store info) */
    int prefix_sum_size = array_size + 1;
    int prefix_sum_byte_size = prefix_sum_size * sizeof(int);

    int* device_prefix_sum;
    hipMalloc(&device_prefix_sum,prefix_sum_byte_size);
    hipMemset(device_prefix_sum,0,prefix_sum_byte_size);

    int* device_ops;
    int ops_byte_size = num_ops * sizeof(std::array<int,3>);
    hipMalloc(&device_ops,ops_byte_size);
    hipMemcpy(device_ops,&ops[0],ops_byte_size,hipMemcpyHostToDevice);

    std::vector<int> host_prefix_sum(prefix_sum_size);
    int threads_per_block = 256;

    int query_offset = 0;
    for(int batch_iter = 0; batch_iter < num_batches; batch_iter++){
        int batch_start = batch_starts[batch_iter];
        int batch_end = (batch_iter == num_batches-1) ? num_ops : batch_starts[batch_iter+1];
        int batch_type = ops[batch_start][0];
        int batch_size = batch_end - batch_start;

        int num_blocks = (batch_size + threads_per_block - 1)/threads_per_block;

        if(batch_type == UPDATE){
            process_update_batch<<<num_blocks,threads_per_block>>>(device_prefix_sum,device_ops,batch_start,batch_size,prefix_sum_size);
            hipDeviceSynchronize();
        }
        else if(batch_type == QUERY){
            hipMemcpy(&host_prefix_sum[0],device_prefix_sum,prefix_sum_byte_size,hipMemcpyDeviceToHost);
            for (int op_i = batch_start; op_i < batch_end; op_i++){
                int l = ops[op_i][1];
                int r = ops[op_i][2];
                int query_answer = host_prefix_sum[r] - host_prefix_sum[l];
                query_results[query_offset] = {op_i,query_answer};
                query_offset++;
            }
        }
    }
    hipFree(device_prefix_sum);
    hipFree(device_ops);
}

